
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
__global__ void vecAdd(double *A, double *B, double *C){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	C[idx] = A[idx] + B[idx];
}

int main(int argc, char* argv[]){
	int n;
	sscanf(argv[1], "%d", &n);
	double *A = (double*)malloc(n * sizeof(double));
	double *B = (double*)malloc(n * sizeof(double));
	double *res = (double*)malloc(n * sizeof(double));

	std::cout << "Please, enter first vector: ";
	for (int i = 0; i < n; ++i){
		std::cin >> A[i];
	}
	std::cout << "Please, enter second vector: ";
	for (int i = 0; i < n; ++i){
		std::cin >> B[i];
	}

	double *A_gpu, *B_gpu, *res_gpu;

	size_t bytes = n * sizeof(double);

	hipMalloc(&A_gpu, bytes);
	hipMalloc(&B_gpu, bytes);
	hipMalloc(&res_gpu, bytes);

	hipMemcpy(A_gpu, A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, bytes, hipMemcpyHostToDevice);

	int block_size = 1024;
	int grid_size = (n - 1) / block_size + 1;

	vecAdd<<<grid_size, block_size>>>(A_gpu, B_gpu, res_gpu);

	hipMemcpy(res, res_gpu, bytes, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(res_gpu);

	for(int i = 0; i < n; ++i){
		std::cout << res[i] << std::endl;
	}
}
